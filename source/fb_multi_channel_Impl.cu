#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <vector>
#include <math.h>
#include ""
#include "fb_multi_channel_Impl.cuh"

__global__ void mupltiply_sum(hipfftComplex* signal, hipfftComplex* resultVec, float* filterTaps, unsigned k,
                                unsigned step, unsigned filterLen, unsigned channelCount, unsigned fftSize, unsigned fftCount)
{
    unsigned sub_batch_size = fftSize / blockDim.x;
    unsigned sub_batch_index = blockIdx.x % sub_batch_size;
    unsigned f_index = sub_batch_index * sub_batch_size + threadIdx.x;
    unsigned batch_index = blockIdx.x / sub_batch_size;
    unsigned index = (batch_index * step + f_index) * channelCount;
    unsigned res_index = batch_index * fftSize + f_index;
    //printf("sub_batch_size %d, sub_batch_index %d, batch_index %d, f_index %d, index %d, res_index %d\n", sub_batch_size, sub_batch_index, batch_index, f_index, index, res_index);
    hipfftComplex result;
    result.x = 0;
    result.y = 0;

    for (unsigned i = 0; i < k; ++i)
    {
        unsigned sig_index = i * fftSize * channelCount + index;
        unsigned h_index = i * fftSize + f_index;
        result.x += filterTaps[h_index] * signal[sig_index].x;
        result.y += filterTaps[h_index] * signal[sig_index].y;
    }

    resultVec[res_index].x = result.x;
    resultVec[res_index].y = result.y;
}



int executeImpl(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen,
                unsigned fftSize, unsigned step, unsigned channelCount, float* result, unsigned long resultLen)
{
    unsigned zerosToPad;
    if (signalLen % filterLen == 0){
        zerosToPad = 0;
    }
    else{
        zerosToPad = filterLen - signalLen % filterLen;
    }
    printf("Zeros to pad: %d\n", zerosToPad);
    unsigned newSignalLen = signalLen + zerosToPad;
    unsigned fftCount = ((newSignalLen - filterLen) / step) + 1;

    hipfftHandle plan;
    hipfftResult cufftStatus;
    cufftStatus = hipfftPlan1d(&plan, fftSize, HIPFFT_C2C, fftCount * channelCount);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftPlan1d failed. Error code %d!\n", cufftStatus);
        return hipErrorUnknown;
    }

    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);


    float* dev_inSignal;
    float* dev_filterTaps;
    hipfftComplex* dev_result;

    unsigned threads_per_block;
    unsigned num_Blocks;

    threads_per_block = 1024;
    num_Blocks = fftCount * ceil((double)fftSize / threads_per_block);
    printf("threads_per_block %d, num_Blocks %d\n", threads_per_block, num_Blocks);
    
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    
    cudaStatus = hipMalloc((float**)&dev_inSignal, sizeof(float) * channelCount * 2 * newSignalLen);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 1\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_filterTaps, filterLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 3\n");
        return cudaStatus;
    }

    float * zeros = new float[2 * zerosToPad * channelCount]();
    cudaStatus = hipMemcpy(dev_inSignal, zeros, 2 * zerosToPad * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 4\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal + channelCount * 2 * zerosToPad, inSignal, 2 * signalLen * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 5\n");
        return cudaStatus;
    }
   
    cudaStatus = hipMemcpy(dev_filterTaps, filterTaps, filterLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 6\n");
        return cudaStatus;
    }


    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);

    for (int channelIndex = 0; channelIndex < channelCount; channelIndex++) {
        mupltiply_sum << <num_Blocks, threads_per_block >> > (dev_inComplexSignal + channelIndex, dev_result + fftCount*fftSize*channelIndex,
            dev_filterTaps, filterLen / fftSize, step, filterLen, channelCount, fftSize, fftCount);
    }

    

    cufftStatus = hipfftExecC2C(plan, dev_result, dev_result, HIPFFT_FORWARD);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
        return hipErrorUnknown;
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_result), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_filterTaps);
    hipFree(dev_result);


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuda execution time (without hipfftPlan1d): %f ms\n", milliseconds);

    return cudaStatus;
}


__inline__ __device__ hipfftComplex operator + (hipfftComplex const& a, hipfftComplex const& b) {
    hipfftComplex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}