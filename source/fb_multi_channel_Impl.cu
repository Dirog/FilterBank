#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <math.h>
#include ""
#include "fb_multi_channel_Impl.cuh"
#include "filterbank.hpp"

__global__ void multiplyAndSum(hipfftComplex* signal, hipfftComplex* resultVec, float* filterTaps,
                            unsigned step, unsigned filterLen, unsigned channelCount, unsigned fftSize, unsigned totalSignalLen, unsigned sub_batch_count)
{
    unsigned sub_batch_index = blockIdx.x % sub_batch_count;
    unsigned h_index = sub_batch_index * blockDim.x + threadIdx.x;
    unsigned f_index = h_index % fftSize;
    unsigned batch_index = blockIdx.x / sub_batch_count;
    unsigned index = (batch_index * step + h_index) * channelCount;
    unsigned res_index = batch_index * fftSize + f_index;

    float tap = filterTaps[h_index];
    for (unsigned i = 0; i < channelCount; ++i)
    {
        unsigned new_res_index = channelCount * res_index + i;
        unsigned signal_index = index + i;

        if(signal_index < totalSignalLen)
        {
            atomicAdd(&(resultVec[new_res_index].x), tap * signal[signal_index].x);
            atomicAdd(&(resultVec[new_res_index].y), tap * signal[signal_index].y);
        }
    }
}

__global__ void multiply(hipfftComplex* tensor, hipfftComplex* factors, unsigned fftCount, unsigned fftSize)
{
    unsigned x  = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned y  = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned k = x % fftCount;
    unsigned f = y % fftSize;

    unsigned tensor_index = x*fftSize + y;
    unsigned factor_index = k*fftSize + f;

    float a = tensor[tensor_index].x;
    float b = tensor[tensor_index].y;
    float c = factors[factor_index].x;
    float d = factors[factor_index].y;
    float re = a*c - b*d;
    float im = b*c + a*d;
    tensor[tensor_index].x = re;
    tensor[tensor_index].y = im;
}

int executeImpl(float* inSignal, unsigned signalLen, float* dev_filterTaps, unsigned filterLen,
                unsigned fftSize, unsigned step, unsigned channelCount, float* result,
                unsigned long resultLen, unsigned threads_per_block, hipfftHandle plan, hipfftComplex* dev_phaseFactors, hipfftComplex* dev_history)
{
printf("resultLen = %d\n", resultLen / 2);

    if (threads_per_block > fftSize){
        threads_per_block = fftSize;
    }

    // unsigned zerosToPad = filterLen - 1;
    // printf("Zeros to pad: %d\n", zerosToPad);
    unsigned historyLen = filterLen - 1;
    unsigned newSignalLen = signalLen + historyLen;
    unsigned fftCount = signalLen / step;

    printf("new matrixSize  = %d\n", newSignalLen*channelCount);

    hipfftResult cufftStatus;

    float* dev_inSignal;
    hipfftComplex* dev_result;
    hipfftComplex* dev_tensor;

    unsigned num_Blocks;
    num_Blocks = fftCount* ceil((double)filterLen / threads_per_block);
    printf("threads_per_block %d, num_Blocks %d\n", threads_per_block, num_Blocks);
    
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    
    cudaStatus = hipMalloc((float**)&dev_inSignal, sizeof(float) * channelCount * 2 * newSignalLen);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 1\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_tensor, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 3\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal, dev_history, 2 * historyLen * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 4\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal + 2 * historyLen * channelCount, inSignal, 2 * signalLen * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 5\n");
        return cudaStatus;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);

    multiplyAndSum << <num_Blocks, threads_per_block >> > (dev_inComplexSignal, dev_result,
                                                          dev_filterTaps, step, filterLen,
                                                          channelCount, fftSize, signalLen*channelCount, filterLen / threads_per_block);

    unsigned threads = (int)sqrt(threads_per_block);
    unsigned blocks = (int)sqrt(resultLen / 2 / threads_per_block);
    dim3 threadsInBlock (threads, threads);
    dim3 blocksPerGrid(blocks, blocks);

    for (int i = 0; i < channelCount; ++i)
    {
        cufftStatus = hipfftExecC2C(plan, dev_result + i, dev_tensor + i*(fftSize*fftCount), HIPFFT_FORWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
            return hipErrorUnknown;
        }
    }
    
    //multiply <<<blocksPerGrid, threadsInBlock>>> (dev_tensor, dev_phaseFactors, fftCount, fftSize);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuda execution time (without cufftPlan and memory operations): %f ms\n", milliseconds);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }


    unsigned endPos = signalLen * channelCount - (filterLen - 1) * channelCount;
    cudaStatus = hipMemcpy(dev_history, dev_inSignal + endPos, (filterLen - 1) * channelCount * sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_tensor), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_result);
    hipFree(dev_tensor);

    return cudaStatus;
}