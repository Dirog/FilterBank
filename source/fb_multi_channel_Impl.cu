#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <math.h>
#include ""
#include "fb_multi_channel_Impl.cuh"
#include "filterbank.hpp"

__inline__ __device__ hipfftComplex operator * (hipfftComplex const& a, hipfftComplex const& b) {
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

__global__ void multiplyAndSum(hipfftComplex* signal, hipfftComplex* resultVec, hipfftComplex* history,
    float* filterTaps, unsigned step, unsigned channelCount, unsigned fftSize, unsigned filterLen,
    unsigned totalSignalLen, unsigned total_historyLen, unsigned sub_batch_count)
{
    unsigned sub_batch_index = blockIdx.x % sub_batch_count;
    unsigned h_index = (sub_batch_index * blockDim.x + threadIdx.x);

    if (h_index < filterLen)
    {
        unsigned f_index = h_index % fftSize;
        unsigned batch_index = blockIdx.x / sub_batch_count;
        unsigned index = (batch_index * step + h_index) * channelCount;
        unsigned res_index = batch_index * fftSize + f_index;


        float tap = filterTaps[h_index];
        for (unsigned i = 0; i < channelCount; ++i)
        {
            unsigned new_res_index = channelCount * res_index + i;
            unsigned signal_index = index + i;

            if (signal_index < total_historyLen)
            {
                atomicAdd(&(resultVec[new_res_index].x), tap * history[signal_index].x);
                atomicAdd(&(resultVec[new_res_index].y), tap * history[signal_index].y);
            }
            else if(signal_index < totalSignalLen + total_historyLen)
            {
                atomicAdd(&(resultVec[new_res_index].x), tap * signal[signal_index - total_historyLen].x);
                atomicAdd(&(resultVec[new_res_index].y), tap * signal[signal_index - total_historyLen].y);
            }
        }
    }
}

__global__ void multiply(hipfftComplex* tensor, hipfftComplex* factors, hipfftComplex* initPhaseFactors, unsigned fftSize,
    unsigned fftCount, unsigned tensorlen)
{
    unsigned index  = threadIdx.x + blockDim.x * blockIdx.x;

    if(index < tensorlen)
    {
        unsigned f = index % fftSize;
        unsigned factor_index = index % (fftCount * fftSize); //total_fftSize
        tensor[index] = tensor[index] * factors[factor_index] * initPhaseFactors[f];
    }
}

__global__ void updateInitPhaseFactors(hipfftComplex* initPhaseFactors, unsigned signalLen, unsigned filterLen, unsigned total_fftSize, unsigned fftSize, unsigned fftCount)
{
    unsigned index  = threadIdx.x + blockDim.x * blockIdx.x;
    if (index < total_fftSize){
        unsigned f = index % fftSize;

        double arg = -2 * M_PI * f * (signalLen) / fftSize;

        hipfftComplex phase;
        phase.x = cos(arg);
        phase.y = sin(arg);

        //printf("%f\n", phase.x);

        initPhaseFactors[f] = initPhaseFactors[f] * phase;
    }
}

int executeImpl(float* dev_inSignal, unsigned signalLen, float* dev_filterTaps, unsigned filterLen, unsigned fftSize,
    unsigned step, unsigned channelCount, float* dev_result, unsigned long resultLen, unsigned threads_per_block,
    unsigned packetIndex, hipfftHandle plan, hipfftComplex* dev_phaseFactors, hipfftComplex* dev_history, hipfftComplex* dev_initPhaseFactors)
{
    unsigned historyLen = filterLen - 1;
    unsigned total_historyLen = historyLen * channelCount;
    unsigned total_signalLen = signalLen * channelCount;
    unsigned fftCount = signalLen / step;
    unsigned total_fftSize = fftCount * fftSize;
    hipfftComplex* dev_tensor;

    unsigned num_Blocks;
    num_Blocks = fftCount * ceil((float)filterLen / threads_per_block);

    hipError_t cudaStatus;
    cudaStatus = hipMallocManaged((hipfftComplex**)&dev_tensor, resultLen * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);
    hipfftComplex* dev_complexResult = reinterpret_cast<hipfftComplex*>(dev_result);

    multiplyAndSum <<<num_Blocks, threads_per_block >>> (dev_inComplexSignal, dev_tensor, dev_history, dev_filterTaps,
        step, channelCount, fftSize, filterLen, total_signalLen, total_historyLen, ceil((float)filterLen / threads_per_block));

    hipfftResult cufftStatus;
    for (int i = 0; i < channelCount; ++i)
    {
        cufftStatus = hipfftExecC2C(plan, dev_tensor + i, dev_complexResult + i * total_fftSize, HIPFFT_FORWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
            return hipErrorUnknown;
        }
    }

    num_Blocks = ceil((float)resultLen / threads_per_block);
    multiply <<<num_Blocks, threads_per_block>>> (dev_complexResult, dev_phaseFactors, dev_initPhaseFactors,
        fftSize, fftCount, resultLen);
    dev_result = reinterpret_cast<float*>(dev_complexResult);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time (without cufftPlan and memory operations): %f ms\n", milliseconds);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed\n");
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    unsigned endPos = total_signalLen - total_historyLen;
    cudaStatus = hipMemcpy(dev_history, dev_inComplexSignal + endPos, total_historyLen * sizeof(hipfftComplex),
        hipMemcpyDeviceToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

                            //Временно
    updateInitPhaseFactors<<<1024*1024,1024>>>(dev_initPhaseFactors, signalLen, filterLen, total_fftSize, fftSize, fftCount);

    hipFree(dev_tensor);

    return cudaStatus;
}
