#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include ""
#include "fb_multi_channel_Impl.cuh"

__global__ void mupltiply_sum(hipfftComplex* signal, hipfftComplex* resultVec, float* filterTaps, int k, int step, int filterLen, int channelCount)
{
    int index = (blockIdx.x * step + threadIdx.x)*channelCount;
    int res_index = blockIdx.x * blockDim.x + threadIdx.x;
    hipfftComplex result;
    result.x = 0;
    result.y = 0;

    for (int i = 0; i < k; ++i)
    {
        int sig_index = i * blockDim.x * channelCount + index;
        result.x += filterTaps[i * blockDim.x + threadIdx.x] * signal[sig_index].x;
        result.y += filterTaps[i * blockDim.x + threadIdx.x] * signal[sig_index].y;
    }

    resultVec[res_index].x = result.x;
    resultVec[res_index].y = result.y;
}


int executeImpl(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen,
                    unsigned fftSize, unsigned step, unsigned channelCount, float* result, unsigned long resultLen)
{
    unsigned zerosToPad;
    if (signalLen % filterLen == 0){
        zerosToPad = 0;
    }
    else{
        zerosToPad = filterLen - signalLen % filterLen;
    }
    printf("zeros to pad: %d\n", zerosToPad);
    unsigned newSignalLen = signalLen + zerosToPad;
    unsigned fftCount = ((newSignalLen - filterLen) / step) + 1;

    hipfftHandle plan;
    hipfftResult cufftStatus;
    cufftStatus = hipfftPlan1d(&plan, fftSize, HIPFFT_C2C, fftCount * channelCount);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftPlan1d failed. Error code %d!\n", cufftStatus);
        return hipErrorUnknown;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    float* dev_inSignal;
    float* dev_filterTaps;
    hipfftComplex* dev_result;
    
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    
    cudaStatus = hipMalloc((float**)&dev_inSignal, sizeof(float) * channelCount * 2 * newSignalLen);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 1\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_filterTaps, filterLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 3\n");
        return cudaStatus;
    }

    float * zeros = new float[2 * zerosToPad * channelCount]();
    cudaStatus = hipMemcpy(dev_inSignal, zeros, 2 * zerosToPad * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 4\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal + channelCount * 2 * zerosToPad, inSignal, 2 * signalLen * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 5\n");
        return cudaStatus;
    }
   
    cudaStatus = hipMemcpy(dev_filterTaps, filterTaps, filterLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 6\n");
        return cudaStatus;
    }



    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);

    for (int channelIndex = 0; channelIndex < channelCount; channelIndex++) {
        mupltiply_sum << <fftCount, fftSize >> > (dev_inComplexSignal + channelIndex, dev_result + fftCount*fftSize*channelIndex,
            dev_filterTaps, filterLen / fftSize, step, filterLen, channelCount);
    }

    

    cufftStatus = hipfftExecC2C(plan, dev_result, dev_result, HIPFFT_FORWARD);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
        return hipErrorUnknown;
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_result), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_filterTaps);
    hipFree(dev_result);


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuda execution time (without hipfftPlan1d): %f ms\n", milliseconds);

    return cudaStatus;
}


__inline__ __device__ hipfftComplex operator + (hipfftComplex const& a, hipfftComplex const& b) {
    hipfftComplex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}