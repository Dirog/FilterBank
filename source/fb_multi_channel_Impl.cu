#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <math.h>
#include ""
#include "fb_multi_channel_Impl.cuh"
#include "filterbank.hpp"

__global__ void multiplyAndSum(hipfftComplex* signal, hipfftComplex* resultVec, float* filterTaps,
                            unsigned step, unsigned filterLen, unsigned channelCount, unsigned fftSize)
{
    unsigned sub_batch_count = filterLen / blockDim.x;
    unsigned sub_batch_index = blockIdx.x % sub_batch_count;
    unsigned h_index = sub_batch_index * blockDim.x + threadIdx.x;
    unsigned f_index = h_index % fftSize;
    unsigned batch_index = blockIdx.x / sub_batch_count;
    unsigned index = (batch_index * step + h_index) * channelCount;
    unsigned res_index = batch_index * fftSize + f_index;


    // float arg = -2*M_PI*step*batch_index / fftSize;
    // float rotateRe = cosf(arg);
    // float rotateIm = sinf(arg);

    float tap = filterTaps[h_index];
    for (unsigned i = 0; i < channelCount; ++i)
    {
        unsigned new_res_index = channelCount * res_index + i;
        atomicAdd(&(resultVec[new_res_index].x), tap * signal[index + i].x);
        atomicAdd(&(resultVec[new_res_index].y), tap * signal[index + i].y);
    }
}



int executeImpl(float* inSignal, unsigned signalLen, float* dev_filterTaps, unsigned filterLen,
                unsigned fftSize, unsigned step, unsigned channelCount, float* result,
                unsigned long resultLen, unsigned threads_per_block, hipfftHandle plan)
{
    if (threads_per_block > fftSize){
        threads_per_block = fftSize;
    }

    unsigned zerosToPad;
    if (signalLen % filterLen == 0){
        zerosToPad = 0;
    }
    else{
        zerosToPad = filterLen - signalLen % filterLen;
    }
    printf("Zeros to pad: %d\n", zerosToPad);
    unsigned newSignalLen = signalLen + zerosToPad;
    unsigned fftCount = ((newSignalLen - filterLen) / step) + 1;

    hipfftResult cufftStatus;

    float* dev_inSignal;
    hipfftComplex* dev_result;
    hipfftComplex* dev_tensor;

    unsigned num_Blocks;
    num_Blocks = fftCount * ceil((double)filterLen / threads_per_block);
    printf("threads_per_block %d, num_Blocks %d\n", threads_per_block, num_Blocks);
    
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    
    cudaStatus = hipMalloc((float**)&dev_inSignal, sizeof(float) * channelCount * 2 * newSignalLen);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 1\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_tensor, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2\n");
        return cudaStatus;
    }

    float * zeros = new float[2 * zerosToPad * channelCount]();
    cudaStatus = hipMemcpy(dev_inSignal, zeros, 2 * zerosToPad * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 4\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal + channelCount * 2 * zerosToPad, inSignal, 2 * signalLen * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 5\n");
        return cudaStatus;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);

    multiplyAndSum << <num_Blocks, threads_per_block >> > (dev_inComplexSignal, dev_result,
                                                          dev_filterTaps, step, filterLen,
                                                          channelCount, fftSize);

    for (int i = 0; i < channelCount; ++i)
    {
        cufftStatus = hipfftExecC2C(plan, dev_result + i, dev_tensor + i*(fftSize*fftCount), HIPFFT_FORWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
            return hipErrorUnknown;
        }
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuda execution time (without cufftPlan and memory operations): %f ms\n", milliseconds);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_tensor), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_result);
    hipFree(dev_tensor);

    return cudaStatus;
}


__inline__ __device__ hipfftComplex operator + (hipfftComplex const& a, hipfftComplex const& b) {
    hipfftComplex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}