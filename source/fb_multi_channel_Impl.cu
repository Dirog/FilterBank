#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <math.h>
#include ""
#include "fb_multi_channel_Impl.cuh"
#include "filterbank.hpp"

__global__ void multiplyAndSum(hipfftComplex* signal, hipfftComplex* resultVec, float* filterTaps,
                            unsigned step, unsigned filterLen, unsigned channelCount, unsigned fftSize, unsigned totalSignalLen, unsigned sub_batch_count)
{
    unsigned sub_batch_index = blockIdx.x % sub_batch_count;
    unsigned h_index = sub_batch_index * blockDim.x + threadIdx.x;
    unsigned f_index = h_index % fftSize;
    unsigned batch_index = blockIdx.x / sub_batch_count;
    unsigned index = (batch_index * step + h_index) * channelCount;
    unsigned res_index = batch_index * fftSize + f_index;

    float tap = filterTaps[h_index];
    for (unsigned i = 0; i < channelCount; ++i)
    {
        unsigned new_res_index = channelCount * res_index + i;
        unsigned signal_index = index + i;

        if(signal_index < totalSignalLen)
        {
            atomicAdd(&(resultVec[new_res_index].x), tap * signal[signal_index].x);
            atomicAdd(&(resultVec[new_res_index].y), tap * signal[signal_index].y);
        }
    }
}

__global__ void multiply(hipfftComplex* tensor, hipfftComplex* factors, unsigned fftCount, unsigned fftSize, unsigned channelCount, unsigned tensorlen)
{
    unsigned index  = threadIdx.x + blockDim.x * blockIdx.x;

    if(index < tensorlen)
    {
        unsigned factor_index = index % fftCount*fftSize;

        float a = tensor[index].x;
        float b = tensor[index].y;
        float c = factors[factor_index].x;
        float d = factors[factor_index].y;
        float re = a*c - b*d;
        float im = b*c + a*d;
        tensor[index].x = re;
        tensor[index].y = im;
    }
}

int executeImpl(float* inSignal, unsigned signalLen, float* dev_filterTaps, unsigned filterLen,
                unsigned fftSize, unsigned step, unsigned channelCount, float* result,
                unsigned long resultLen, unsigned threads_per_block, hipfftHandle plan, hipfftComplex* dev_phaseFactors, hipfftComplex* dev_history)
{
    if (threads_per_block > fftSize){
        threads_per_block = fftSize;
    }

    unsigned historyLen = filterLen - 1;
    unsigned newSignalLen = signalLen + historyLen;
    unsigned fftCount = signalLen / step;

    hipfftResult cufftStatus;

    float* dev_inSignal;
    hipfftComplex* dev_result;
    hipfftComplex* dev_tensor;

    unsigned num_Blocks;
    num_Blocks = fftCount * ceil((double)filterLen / threads_per_block);

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    
    cudaStatus = hipMalloc((float**)&dev_inSignal, sizeof(float) * channelCount * 2 * newSignalLen);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 1\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_tensor, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 3\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal, dev_history, 2 * historyLen * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 4\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal + 2 * historyLen * channelCount, inSignal, 2 * signalLen * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 5\n");
        return cudaStatus;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);

    multiplyAndSum << <num_Blocks, threads_per_block >> > (dev_inComplexSignal, dev_result,
                                                          dev_filterTaps, step, filterLen,
                                                          channelCount, fftSize, signalLen*channelCount, filterLen / threads_per_block);

    for (int i = 0; i < channelCount; ++i)
    {
        cufftStatus = hipfftExecC2C(plan, dev_result + i, dev_tensor + i*(fftSize*fftCount), HIPFFT_FORWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
            return hipErrorUnknown;
        }
    }

    num_Blocks = ceil(0.5 * resultLen / threads_per_block);

    multiply <<<num_Blocks, threads_per_block>>> (dev_tensor, dev_phaseFactors, fftCount, fftSize, channelCount, resultLen/2);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuda execution time (without cufftPlan and memory operations): %f ms\n", milliseconds);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }


    unsigned endPos = signalLen * channelCount - (filterLen - 1) * channelCount;
    cudaStatus = hipMemcpy(dev_history, dev_inSignal + endPos, (filterLen - 1) * channelCount * sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_tensor), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_result);
    hipFree(dev_tensor);

    return cudaStatus;
}