#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>
#include <stdio.h>
#include "filterbank.hpp"

__inline__ __device__ hipfftComplex operator * (hipfftComplex const& a, hipfftComplex const& b) {
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

__global__ void multiplyAndSum(hipfftComplex* signal, hipfftComplex* resultVec, hipfftComplex* history,
    float* filterTaps, unsigned step, unsigned channelCount, unsigned fftSize, unsigned filterLen,
    unsigned totalSignalLen, unsigned totalHistoryLen, unsigned subBatchCount)
{
    unsigned sub_batch_index = blockIdx.x % subBatchCount;
    unsigned h_index = (sub_batch_index * blockDim.x + threadIdx.x);

    if (h_index < filterLen)
    {
        unsigned f_index = h_index % fftSize;
        unsigned batch_index = blockIdx.x / subBatchCount;
        unsigned index = (batch_index * step + h_index) * channelCount;
        unsigned res_index = batch_index * fftSize + f_index;


        float tap = filterTaps[h_index];
        for (unsigned i = 0; i < channelCount; ++i)
        {
            unsigned new_res_index = channelCount * res_index + i;
            unsigned signal_index = index + i;

            if (signal_index < totalHistoryLen)
            {
                atomicAdd(&(resultVec[new_res_index].x), tap * history[signal_index].x);
                atomicAdd(&(resultVec[new_res_index].y), tap * history[signal_index].y);
            }
            else if(signal_index < totalSignalLen + totalHistoryLen)
            {
                atomicAdd(&(resultVec[new_res_index].x), tap * signal[signal_index - totalHistoryLen].x);
                atomicAdd(&(resultVec[new_res_index].y), tap * signal[signal_index - totalHistoryLen].y);
            }
        }
    }
}

__global__ void multiply(hipfftComplex* tensor, hipfftComplex* factors, hipfftComplex* initPhaseFactors, unsigned fftSize,
    unsigned fftCount, unsigned tensorlen) //Doesn't work
{
    unsigned index  = threadIdx.x + blockDim.x * blockIdx.x;
    if(index < tensorlen)
    {
        unsigned f = index % fftSize;
        unsigned factor_index = index % (fftCount * fftSize);
        tensor[index] = tensor[index] * factors[factor_index] * initPhaseFactors[f];
    }
}

__global__ void multiplyTest(hipfftComplex* tensor, hipfftComplex* factors, hipfftComplex* initPhaseFactors, unsigned fftSize,
    unsigned fftCount, unsigned tensorlen) //Works
{
    for (int i = 0; i < tensorlen; ++i)
    {
        unsigned f = i % fftSize;
        unsigned factor_index = i % (fftCount * fftSize);
        tensor[i] = tensor[i] * factors[factor_index] * initPhaseFactors[f];
    }

}

__global__ void updateInitPhaseFactors(hipfftComplex* initPhaseFactors, unsigned signalLen, unsigned fftSize)
{
    unsigned index  = threadIdx.x + blockDim.x * blockIdx.x;
    if (index < fftSize){
        unsigned f = index;

        double arg = -2 * M_PI * f * signalLen / fftSize;
        hipfftComplex phase;
        phase.x = cos(arg);
        phase.y = sin(arg);
        initPhaseFactors[f] = initPhaseFactors[f] * phase;
    }
}

int executeImpl(float* dev_inSignal, unsigned signalLen, float* dev_filterTaps, unsigned filterLen, unsigned fftSize,
    unsigned step, unsigned channelCount, float* dev_result, unsigned long resultLen, unsigned threads_per_block,
    hipfftHandle plan, hipfftComplex* dev_phaseFactors, hipfftComplex* dev_history, hipfftComplex* dev_initPhaseFactors)
{
    unsigned historyLen = filterLen - 1;
    unsigned total_historyLen = historyLen * channelCount;
    unsigned total_signalLen = signalLen * channelCount;
    unsigned fftCount = signalLen / step;
    unsigned total_fftSize = fftCount * fftSize;
    hipfftComplex* dev_tensor;

    unsigned num_Blocks;
    num_Blocks = fftCount * ceil((float)filterLen / threads_per_block);

    hipError_t cudaStatus;
    cudaStatus = hipMallocManaged((hipfftComplex**)&dev_tensor, resultLen * sizeof(hipfftComplex));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);
    hipfftComplex* dev_complexResult = reinterpret_cast<hipfftComplex*>(dev_result);

    multiplyAndSum <<<num_Blocks, threads_per_block >>> (dev_inComplexSignal, dev_tensor, dev_history, dev_filterTaps,
        step, channelCount, fftSize, filterLen, total_signalLen, total_historyLen, ceil((float)filterLen / threads_per_block));

    hipfftResult cufftStatus;
    for (int i = 0; i < channelCount; ++i)
    {
        cufftStatus = hipfftExecC2C(plan, dev_tensor + i, dev_complexResult + i * total_fftSize, HIPFFT_FORWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
            return hipErrorUnknown;
        }
    }

    num_Blocks = ceil((float)resultLen / threads_per_block);
     // multiply <<<num_Blocks, threads_per_block>>> (dev_complexResult, dev_phaseFactors, dev_initPhaseFactors,
     //     fftSize, fftCount, resultLen);

    multiply <<<1, 1>>> (dev_complexResult, dev_phaseFactors, dev_initPhaseFactors, fftSize, fftCount, resultLen);

    dev_result = reinterpret_cast<float*>(dev_complexResult);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time (without cufftPlan and memory operations): %f ms\n", milliseconds);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed\n");
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    unsigned endPos = total_signalLen - total_historyLen;
    cudaStatus = hipMemcpy(dev_history, dev_inComplexSignal + endPos, total_historyLen * sizeof(hipfftComplex),
        hipMemcpyDeviceToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    num_Blocks = ceil((float)fftSize / threads_per_block);
    updateInitPhaseFactors<<<num_Blocks,threads_per_block>>>(dev_initPhaseFactors, signalLen, fftSize);
    
    hipFree(dev_tensor);

    return cudaStatus;
}
