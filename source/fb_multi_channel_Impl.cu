#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <math.h>
#include ""
#include "fb_multi_channel_Impl.cuh"

__global__ void multiply_sum(hipfftComplex* signal, hipfftComplex* resultVec, float* filterTaps,
                            unsigned step, unsigned filterLen, unsigned channelCount, unsigned fftSize)
{
    unsigned sub_batch_count = filterLen / blockDim.x;
    unsigned sub_batch_index = blockIdx.x % sub_batch_count;
    unsigned h_index = sub_batch_index * blockDim.x + threadIdx.x;
    unsigned f_index = h_index % fftSize;
    unsigned batch_index = blockIdx.x / sub_batch_count;
    unsigned index = (batch_index * step + h_index) * channelCount;
    unsigned res_index = batch_index * fftSize + f_index;

    float tap = filterTaps[h_index];
    for (unsigned i = 0; i < channelCount; ++i)
    {
        unsigned new_res_index = channelCount * res_index + i;
        atomicAdd(&(resultVec[new_res_index].x), tap * signal[index + i].x);
        atomicAdd(&(resultVec[new_res_index].y), tap * signal[index + i].y);
    }
}



int executeImpl(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen,
                unsigned fftSize, unsigned step, unsigned channelCount, float* result,
                unsigned long resultLen, unsigned threads_per_block)
{
    if (threads_per_block > fftSize){
        threads_per_block = fftSize;
    }

    unsigned zerosToPad;
    if (signalLen % filterLen == 0){
        zerosToPad = 0;
    }
    else{
        zerosToPad = filterLen - signalLen % filterLen;
    }
    printf("Zeros to pad: %d\n", zerosToPad);
    unsigned newSignalLen = signalLen + zerosToPad;
    unsigned fftCount = ((newSignalLen - filterLen) / step) + 1;

    hipfftHandle plan;
    hipfftResult cufftStatus;
    int * nx = new int(fftSize);
    int idist = channelCount * fftSize;
    int odist = fftSize;
    int istride = channelCount, ostride = 1;
    int *inembed = new int(resultLen);
    int *onembed = new int(resultLen);
    cufftStatus = hipfftPlanMany(&plan, 1, nx, inembed, istride, idist, onembed, ostride, odist, HIPFFT_C2C, fftCount);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftPlanMany failed. Error code %d!\n", cufftStatus);
        return hipErrorUnknown;
    }

    float* dev_inSignal;
    float* dev_filterTaps;
    hipfftComplex* dev_result;
    hipfftComplex* dev_tensor;

    unsigned num_Blocks;
    num_Blocks = fftCount * ceil((double)filterLen / threads_per_block);
    printf("threads_per_block %d, num_Blocks %d\n", threads_per_block, num_Blocks);
    
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    
    cudaStatus = hipMalloc((float**)&dev_inSignal, sizeof(float) * channelCount * 2 * newSignalLen);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 1\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_tensor, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_filterTaps, filterLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 3\n");
        return cudaStatus;
    }

    float * zeros = new float[2 * zerosToPad * channelCount]();
    cudaStatus = hipMemcpy(dev_inSignal, zeros, 2 * zerosToPad * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 4\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal + channelCount * 2 * zerosToPad, inSignal, 2 * signalLen * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 5\n");
        return cudaStatus;
    }
   
    cudaStatus = hipMemcpy(dev_filterTaps, filterTaps, filterLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 6\n");
        return cudaStatus;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);

    multiply_sum << <num_Blocks, threads_per_block >> > (dev_inComplexSignal, dev_result,
                                                          dev_filterTaps, step, filterLen,
                                                          channelCount, fftSize);

    for (int i = 0; i < channelCount; ++i)
    {
        cufftStatus = hipfftExecC2C(plan, dev_result + i, dev_tensor + i*(fftSize*fftCount), HIPFFT_FORWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
            return hipErrorUnknown;
        }
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuda execution time (without cufftPlan and memory operations): %f ms\n", milliseconds);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_tensor), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_filterTaps);
    hipFree(dev_result);
    hipFree(dev_tensor);

    return cudaStatus;
}


__inline__ __device__ hipfftComplex operator + (hipfftComplex const& a, hipfftComplex const& b) {
    hipfftComplex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}