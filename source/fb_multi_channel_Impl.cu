#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <math.h>
#include ""
#include "fb_multi_channel_Impl.cuh"
#include "filterbank.hpp"

__global__ void multiplyAndSum(hipfftComplex* signal, hipfftComplex* resultVec, float* filterTaps,
                            unsigned step, unsigned filterLen, unsigned channelCount, unsigned fftSize)
{
    unsigned sub_batch_count = filterLen / blockDim.x;
    unsigned sub_batch_index = blockIdx.x % sub_batch_count;
    unsigned h_index = sub_batch_index * blockDim.x + threadIdx.x;
    unsigned f_index = h_index % fftSize;
    unsigned batch_index = blockIdx.x / sub_batch_count;
    unsigned index = (batch_index * step + h_index) * channelCount;
    unsigned res_index = batch_index * fftSize + f_index;

    float tap = filterTaps[h_index];
    for (unsigned i = 0; i < channelCount; ++i)
    {
        unsigned new_res_index = channelCount * res_index + i;
        atomicAdd(&(resultVec[new_res_index].x), tap * signal[index + i].x);
        atomicAdd(&(resultVec[new_res_index].y), tap * signal[index + i].y);
    }
}

__global__ void multiply(hipfftComplex* tensor, hipfftComplex* factors, unsigned fftCount, unsigned fftSize)
{
    unsigned k  = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned f  = threadIdx.y + blockDim.y * blockIdx.y;

    if(k < fftCount && f < fftSize)
    {
        unsigned tensor_index = k*fftSize + f;
        unsigned factor_index = k*fftSize + f;

        float a = tensor[tensor_index].x;
        float b = tensor[tensor_index].y;
        float c = factors[factor_index].x;
        float d = factors[factor_index].y;
        float re = a*c - b*d;
        float im = b*c + a*d;
        tensor[tensor_index].x = re;
        tensor[tensor_index].y = im;
    }
}

int executeImpl(float* inSignal, unsigned signalLen, float* dev_filterTaps, unsigned filterLen,
                unsigned fftSize, unsigned step, unsigned channelCount, float* result,
                unsigned long resultLen, unsigned threads_per_block, hipfftHandle plan, hipfftComplex* dev_phaseFactors)
{
    if (threads_per_block > fftSize){
        threads_per_block = fftSize;
    }

    unsigned zerosToPad = filterLen - 1;
    printf("Zeros to pad: %d\n", zerosToPad);
    unsigned newSignalLen = signalLen + zerosToPad;
    unsigned fftCount = signalLen / step;

    hipfftResult cufftStatus;

    float* dev_inSignal;
    hipfftComplex* dev_result;
    hipfftComplex* dev_tensor;

    unsigned num_Blocks;
    num_Blocks = fftCount * ceil((double)filterLen / threads_per_block);
    printf("threads_per_block %d, num_Blocks %d\n", threads_per_block, num_Blocks);
    
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    
    cudaStatus = hipMalloc((float**)&dev_inSignal, sizeof(float) * channelCount * 2 * newSignalLen);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 1\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_tensor, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 3\n");
        return cudaStatus;
    }

    float * zeros = new float[2 * zerosToPad * channelCount]();
    cudaStatus = hipMemcpy(dev_inSignal, zeros, 2 * zerosToPad * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 4\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal + channelCount * 2 * zerosToPad, inSignal, 2 * signalLen * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 5\n");
        return cudaStatus;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);

    multiplyAndSum << <num_Blocks, threads_per_block >> > (dev_inComplexSignal, dev_result,
                                                          dev_filterTaps, step, filterLen,
                                                          channelCount, fftSize);

    dim3 threadsInBlock (32, 32);
    dim3 blocksPerGrid(1024, 1024); //Временно

    for (int i = 0; i < channelCount; ++i)
    {
        cufftStatus = hipfftExecC2C(plan, dev_result + i, dev_tensor + i*(fftSize*fftCount), HIPFFT_FORWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
            return hipErrorUnknown;
        }

        multiply <<<blocksPerGrid, threadsInBlock>>> (dev_tensor + i*fftSize*fftCount, dev_phaseFactors, fftCount, fftSize);
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuda execution time (without cufftPlan and memory operations): %f ms\n", milliseconds);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_tensor), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_result);
    hipFree(dev_tensor);

    return cudaStatus;
}