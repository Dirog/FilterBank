#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <math.h>
#include ""
#include "fb_multi_channel_Impl.cuh"
#include "filterbank.hpp"

__global__ void multiplyAndSum(hipfftComplex* signal, hipfftComplex* resultVec,
    float* filterTaps, unsigned step, unsigned channelCount, unsigned fftSize,
    unsigned totalSignalLen, unsigned sub_batch_count)
{
    unsigned sub_batch_index = blockIdx.x % sub_batch_count;
    unsigned h_index = sub_batch_index * blockDim.x + threadIdx.x;
    unsigned f_index = h_index % fftSize;
    unsigned batch_index = blockIdx.x / sub_batch_count;
    unsigned index = (batch_index * step + h_index) * channelCount;
    unsigned res_index = batch_index * fftSize + f_index;

    float tap = filterTaps[h_index];
    for (unsigned i = 0; i < channelCount; ++i)
    {
        unsigned new_res_index = channelCount * res_index + i;
        unsigned signal_index = index + i;

        if(signal_index < totalSignalLen)
        {
            atomicAdd(&(resultVec[new_res_index].x), tap * signal[signal_index].x);
            atomicAdd(&(resultVec[new_res_index].y), tap * signal[signal_index].y);
        }
    }
}

__global__ void multiply(hipfftComplex* tensor, hipfftComplex* factors,
    unsigned total_fftSize, unsigned tensorlen)
{
    unsigned index  = threadIdx.x + blockDim.x * blockIdx.x;

    if(index < tensorlen)
    {
        unsigned factor_index = index % total_fftSize;

        float a = tensor[index].x;
        float b = tensor[index].y;
        float c = factors[factor_index].x;
        float d = factors[factor_index].y;
        tensor[index].x = a*c - b*d;
        tensor[index].y = b*c + a*d;
    }
}

int executeImpl(float* inSignal, unsigned signalLen, float* dev_filterTaps, unsigned filterLen,
    unsigned fftSize, unsigned step, unsigned channelCount, float* result, unsigned long resultLen,
    unsigned threads_per_block, hipfftHandle plan, hipfftComplex* dev_phaseFactors, hipfftComplex* dev_history)
{
    if (threads_per_block > fftSize){
        threads_per_block = fftSize;
    }

    unsigned historyLen = filterLen - 1;
    unsigned newSignalLen = signalLen + historyLen;
    unsigned total_historyLen = historyLen * channelCount;
    unsigned total_signalLen = signalLen * channelCount;
    unsigned fftCount = signalLen / step;
    unsigned total_fftSize = fftCount * fftSize;

    float* dev_inSignal;
    hipfftComplex* dev_result;
    hipfftComplex* dev_tensor;

    unsigned num_Blocks;
    num_Blocks = fftCount * ceil((double)filterLen / threads_per_block);

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_inSignal, 2 * newSignalLen * channelCount * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 1\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_result, 2 * resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 2\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_tensor, 2 * resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed! 3\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal, dev_history, 2 * total_historyLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 4\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal + 2 * total_historyLen, inSignal, 2 * total_signalLen * sizeof(float),
        hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 5\n");
        return cudaStatus;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);

    multiplyAndSum <<<num_Blocks, threads_per_block >>> (dev_inComplexSignal, dev_result,dev_filterTaps,
        step, channelCount, fftSize, total_signalLen, filterLen / threads_per_block);

    hipfftResult cufftStatus;
    for (int i = 0; i < channelCount; ++i)
    {
        cufftStatus = hipfftExecC2C(plan, dev_result + i, dev_tensor + i * total_fftSize, HIPFFT_FORWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
            return hipErrorUnknown;
        }
    }

    num_Blocks = ceil(resultLen / threads_per_block);
    multiply <<<num_Blocks, threads_per_block>>> (dev_tensor, dev_phaseFactors, total_fftSize, resultLen);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time (without cufftPlan and memory operations): %f ms\n", milliseconds);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed\n");
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    unsigned endPos = total_signalLen - total_historyLen;
    cudaStatus = hipMemcpy(dev_history, dev_inSignal + endPos, total_historyLen * sizeof(hipfftComplex),
        hipMemcpyDeviceToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 6\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_tensor), 2 * resultLen * sizeof(float),
        hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed! 7\n");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_result);
    hipFree(dev_tensor);

    return cudaStatus;
}