#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include ""
#include "../include/fb_multi_channel_Impl.cuh"

__global__ void mupltiply_sum(hipfftComplex* signal, hipfftComplex* resultVec, float* filterTaps, int k, int step, int filterLen, int channelCount)
{
    int index = (blockIdx.x * step + threadIdx.x)*channelCount;
    int res_index = blockIdx.x * blockDim.x + threadIdx.x;
    hipfftComplex result;
    result.x = 0;
    result.y = 0;

    for (int i = 0; i < k; ++i)
    {
        int sig_index = i * blockDim.x * channelCount + index;
        result.x += filterTaps[i * blockDim.x + threadIdx.x] * signal[sig_index].x;
        result.y += filterTaps[i * blockDim.x + threadIdx.x] * signal[sig_index].y;
    }

    resultVec[res_index].x = result.x;
    resultVec[res_index].y = result.y;
}


int executeImpl(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen,
                    unsigned fftSize, unsigned step, unsigned channelCount, float* result, unsigned long resultLen)
{
    unsigned fftCount = ((signalLen / 2 - filterLen) / step) + 1;
    hipfftHandle plan;
    hipfftResult cufftStatus;
    cufftStatus = hipfftPlan1d(&plan, fftSize, HIPFFT_C2C, fftCount * channelCount);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftPlan1d failed. Error code %d!\n", cufftStatus);
        return hipErrorUnknown;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    float* dev_inSignal;
    float* dev_filterTaps;
    hipfftComplex* dev_result;
    
    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_inSignal, signalLen * channelCount * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMallocManaged((float**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_filterTaps, filterLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }


    cudaStatus = hipMemcpy(dev_inSignal, inSignal, signalLen * channelCount * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_filterTaps, filterTaps, filterLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);

    for (int channelIndex = 0; channelIndex < channelCount; channelIndex++) {
        mupltiply_sum << <fftCount, fftSize >> > (dev_inComplexSignal + channelIndex, dev_result + fftCount*fftSize*channelIndex,
            dev_filterTaps, filterLen / fftSize, step, filterLen, channelCount);
    }

    

    cufftStatus = hipfftExecC2C(plan, dev_result, dev_result, HIPFFT_FORWARD);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
        return hipErrorUnknown;
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_result), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_filterTaps);
    hipFree(dev_result);


    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("cuda execution time (without hipfftPlan1d): %f ms\n", milliseconds);

    return cudaStatus;
}


__inline__ __device__ hipfftComplex operator + (hipfftComplex const& a, hipfftComplex const& b) {
    hipfftComplex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}