#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <math.h>
#include ""
#include "fb_multi_channel_Impl.cuh"
#include "filterbank.hpp"

__inline__ __device__ hipfftComplex operator * (hipfftComplex const& a, hipfftComplex const& b) {
    hipfftComplex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

__global__ void multiplyAndSum(hipfftComplex* signal, hipfftComplex* resultVec, hipfftComplex* history,
    float* filterTaps, unsigned step, unsigned channelCount, unsigned fftSize,
    unsigned totalSignalLen, unsigned total_historyLen, unsigned sub_batch_count)
{
    unsigned sub_batch_index = blockIdx.x % sub_batch_count;
    unsigned h_index = (sub_batch_index * blockDim.x + threadIdx.x);
    unsigned f_index = h_index % fftSize;
    unsigned batch_index = blockIdx.x / sub_batch_count;
    unsigned index = (batch_index * step + h_index) * channelCount;
    unsigned res_index = batch_index * fftSize + f_index;

    float tap = filterTaps[h_index];
    for (unsigned i = 0; i < channelCount; ++i)
    {
        unsigned new_res_index = channelCount * res_index + i;
        unsigned signal_index = index + i;

        if (signal_index < total_historyLen)
        {
            atomicAdd(&(resultVec[new_res_index].x), tap * history[signal_index].x);
            atomicAdd(&(resultVec[new_res_index].y), tap * history[signal_index].y);
        }
        else if(signal_index < totalSignalLen + total_historyLen)
        {
            atomicAdd(&(resultVec[new_res_index].x), tap * signal[signal_index - total_historyLen].x);
            atomicAdd(&(resultVec[new_res_index].y), tap * signal[signal_index - total_historyLen].y);
        }
    }
}

__global__ void multiply(hipfftComplex* tensor, hipfftComplex* factors, unsigned fftSize,
    unsigned fftCount, unsigned tensorlen, unsigned packetIndex, unsigned signalLen, unsigned filterLen)
{
    unsigned index  = threadIdx.x + blockDim.x * blockIdx.x;

    if(index < tensorlen)
    {
        unsigned factor_index = index % (fftCount * fftSize);
        tensor[index] = tensor[index] * factors[factor_index];
    }
}

int executeImpl(float* dev_inSignal, unsigned signalLen, float* dev_filterTaps, unsigned filterLen, unsigned fftSize,
    unsigned step, unsigned channelCount, float* dev_result, unsigned long resultLen,unsigned threads_per_block,
    unsigned packetIndex, hipfftHandle plan, hipfftComplex* dev_phaseFactors, hipfftComplex* dev_history)
{
    if (threads_per_block > fftSize){
        threads_per_block = fftSize;
    }

    unsigned historyLen = filterLen - 1;
    unsigned total_historyLen = historyLen * channelCount;
    unsigned total_signalLen = signalLen * channelCount;
    unsigned fftCount = signalLen / step;
    unsigned total_fftSize = fftCount * fftSize;
    hipfftComplex* dev_tensor;

    unsigned num_Blocks;
    num_Blocks = (fftCount) * ceil((float)filterLen / threads_per_block);

    hipError_t cudaStatus;
    cudaStatus = hipMalloc((float**)&dev_tensor, 2 * resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);
    hipfftComplex* dev_complexResult = reinterpret_cast<hipfftComplex*>(dev_result);

        printf("%f, %f\n", dev_inComplexSignal[total_signalLen - 1].x, dev_inComplexSignal[total_signalLen - 1].y);

    multiplyAndSum <<<num_Blocks, threads_per_block >>> (dev_inComplexSignal, dev_tensor, dev_history, dev_filterTaps,
        step, channelCount, fftSize, total_signalLen, total_historyLen, ceil((float)filterLen / threads_per_block));

    hipfftResult cufftStatus;
    for (int i = 0; i < channelCount; ++i)
    {
        cufftStatus = hipfftExecC2C(plan, dev_tensor + i, dev_complexResult + i * total_fftSize, HIPFFT_FORWARD);
        if (cufftStatus != HIPFFT_SUCCESS) {
            fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
            return hipErrorUnknown;
        }
    }

    num_Blocks = ceil((float)resultLen / threads_per_block);
    multiply <<<num_Blocks, threads_per_block>>> (dev_complexResult, dev_phaseFactors,
        fftSize, fftCount, resultLen, packetIndex, signalLen, filterLen);
    dev_result = reinterpret_cast<float*>(dev_complexResult);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time (without cufftPlan and memory operations): %f ms\n", milliseconds);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed\n");
        return cudaStatus;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    unsigned endPos = total_signalLen - total_historyLen;
    cudaStatus = hipMemcpy(dev_history, dev_inComplexSignal + endPos, total_historyLen * sizeof(hipfftComplex),
        hipMemcpyDeviceToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipFree(dev_tensor);

    return cudaStatus;
}