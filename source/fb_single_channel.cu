#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include ""

hipError_t execute(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, unsigned fftSize, unsigned step, float* result, unsigned resultLen);
void readVectorFromFile(const char* fileName, float* result, int len);
void writeResultToFile(const char* fileName, float* result, int x, int y);
__inline__ __device__ hipfftComplex operator + (hipfftComplex const& a, hipfftComplex const& b);

__global__ void mupltiply_sum(hipfftComplex* signal, hipfftComplex* resultVec, float* filterTaps, int k, int step)
{
    int index = blockIdx.x * step + threadIdx.x;
    int res_index = blockIdx.x * blockDim.x + threadIdx.x;
    hipfftComplex result;
    result.x = 0;
    result.y = 0;

    for (int i = 0; i < k; ++i)
    {
        result.x += filterTaps[i * blockDim.x + threadIdx.x] * signal[i * blockDim.x + index].x;
        result.y += filterTaps[i * blockDim.x + threadIdx.x] * signal[i * blockDim.x + index].y;
    }

    resultVec[res_index] = result;
}



int main() {
    const int signalLen = 1024 * 8 * 2;
    const int filterLen = 128;
    const int fftSize = filterLen / 16;
    const int step = 32;
    const int fftCount = ((signalLen / 2 - filterLen) / step) + 1;
    const int resultLen = 2 * fftSize * fftCount;
    float* result = new float[resultLen];

    float inSignal[signalLen];
    float filterTaps[filterLen];

    readVectorFromFile("../python/files/signal", inSignal, signalLen);
    readVectorFromFile("../python/files/taps", filterTaps, filterLen);

    hipError_t cudaStatus;
    cudaStatus = execute(inSignal, signalLen, filterTaps, filterLen, fftSize, step, result, resultLen);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Execution failed\n");
        return -1;
    }

    writeResultToFile("../python/files/result", result, 2 * fftSize, fftCount);

    return 0;
}

hipError_t execute(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, const unsigned fftSize, unsigned step, float* result, unsigned resultLen)
{
    float* dev_inSignal;
    float* dev_filterTaps;
    hipfftComplex* dev_result;

    int fftCount = ((signalLen / 2 - filterLen) / step) + 1;

    hipError_t cudaStatus;
    hipfftResult cufftStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_inSignal, signalLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMallocManaged((float**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_filterTaps, filterLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal, inSignal, signalLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_filterTaps, filterTaps, filterLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);
   
    mupltiply_sum << <fftCount, fftSize >> > (dev_inComplexSignal, dev_result, dev_filterTaps, filterLen / fftSize, step);

    hipfftHandle plan;
    cufftStatus = hipfftPlan1d(&plan, fftSize, HIPFFT_C2C, fftCount);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftPlan1d failed. Error code %d!\n", cufftStatus);
        return hipErrorUnknown;
    }

    cufftStatus = hipfftExecC2C(plan, dev_result,
        dev_result, HIPFFT_FORWARD);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
        return hipErrorUnknown;
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_result), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_filterTaps);
    hipFree(dev_result);

    return cudaStatus;
}


__device__ hipfftComplex operator + (hipfftComplex const& a, hipfftComplex const& b) {
    hipfftComplex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

void readVectorFromFile(const char* fileName, float* result, int len) {
    FILE* file;
    file = fopen(fileName, "r");
    if (file == NULL) {
        printf("Error reading file!\n");
        return;
    }
    for (int m = 0; m < len; ++m) {
        fscanf(file, "%f ", &result[m]);
    }
    fclose(file);
}

void writeResultToFile(const char* fileName, float* result, int x, int y) {
    FILE* file;
    file = fopen(fileName, "w");

    int  n = 0;
    for (int l = 0; l < y; ++l) {
        for (int i = 0; i < x; ++i) {
            fprintf(file, "%f ", result[n]);
            n++;
        }
        fprintf(file, "\n");
    }
    fclose(file);
}