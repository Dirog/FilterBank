#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include ""

#define DIM 1024

hipError_t execute(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, unsigned fftSize, unsigned step, float* result, unsigned resultLen);
void readVectorFromFile(const char* fileName, float* result, int len);
void writeResultToFile(const char* fileName, float* result, int x, int y);
__device__ hipfftComplex operator + (hipfftComplex const& a, hipfftComplex const& b);


__global__ void multiply(hipfftComplex* signalVec, float* filterTaps, hipfftComplex* vecOut, int size, int step, int stepIndx)
{
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < size / step) {
        int indx = step * i + stepIndx;
        vecOut[i].x = signalVec[indx].x * filterTaps[indx];
        vecOut[i].y = signalVec[indx].y * filterTaps[indx];
    }
}

__global__ void reductionSum(hipfftComplex* vect, hipfftComplex* vecOut, int size, int fftSize, int stepIndx)
{
    __shared__ hipfftComplex block[DIM];
    unsigned int globalIndex = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int i = threadIdx.x;

    if (globalIndex < size) {
        block[i] = vect[globalIndex];
    }
    else {
        block[i].x = 0;
        block[i].y = 0;
    }

    __syncthreads();

    for (unsigned int j = blockDim.x / 2; j > 0; j >>= 1)
    {
        if (i < j){
            block[i] = block[i] + block[i + j];
        }

        __syncthreads();
    }

    if (i == 0){
        vecOut[blockIdx.x] = block[0];
    }

}


int main() {
    const int signalLen = 1024*8*2;
    const int filterLen = 128;
    const int fftSize = filterLen / 16;
    const int step = 32;
    const int fftCount = ((signalLen / 2 - filterLen) / step) + 1;
    const int resultLen = 2 * fftSize * fftCount;
    float* result = new float[resultLen];

    float inSignal[signalLen];
    float filterTaps[filterLen];

    readVectorFromFile("../python/files/signal", inSignal, signalLen);
    readVectorFromFile("../python/files/taps", filterTaps, filterLen);

    hipError_t cudaStatus;
    cudaStatus = execute(inSignal, signalLen, filterTaps, filterLen, fftSize, step, result, resultLen);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Execution failed\n");
        return -1;
    }

    writeResultToFile("../python/files/result", result, 2 * fftSize, fftCount);

    return 0;
}

hipError_t execute(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, const unsigned fftSize, unsigned step, float* result, unsigned resultLen)
{
    int threadsPerBlock = DIM;

    float* dev_inSignal;
    float* dev_filterTaps;
    hipfftComplex* dev_result;
    hipfftComplex* dev_subVec;
    hipfftComplex* dev_vecOut;

    int fftCount = ((signalLen / 2 - filterLen) / step) + 1;
    int subVecSize = filterLen / fftSize;
    
    hipError_t cudaStatus;
    hipfftResult cufftStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_inSignal, signalLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMallocManaged((float**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_filterTaps, filterLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_subVec, filterLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMallocManaged((float**)&dev_vecOut, subVecSize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal, inSignal, signalLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_filterTaps, filterTaps, filterLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);
    int numInputElements = subVecSize;
    int numOutputElements;
    
    for (int batchIndx = 0; batchIndx < fftCount; batchIndx++)
    {
        for (int stepIndx = 0; stepIndx < fftSize; stepIndx++) {
            multiply << <256, threadsPerBlock >> > (dev_inComplexSignal + batchIndx*step, dev_filterTaps, dev_subVec, filterLen, fftSize, stepIndx);
            do
            {
                numOutputElements = numInputElements / (threadsPerBlock);
                if (numInputElements % (threadsPerBlock)) {
                    numOutputElements++;
                }

                reductionSum << < numOutputElements, threadsPerBlock >> > (dev_subVec, dev_vecOut, numInputElements, fftSize, stepIndx);
                numInputElements = numOutputElements;
                if (numOutputElements > 1) {
                    reductionSum << < numOutputElements, threadsPerBlock >> > (dev_vecOut, dev_subVec, numInputElements, fftSize, stepIndx);
                }

            } while (numOutputElements > 1);

            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
                return cudaStatus;
            }

            dev_result[stepIndx + batchIndx*fftSize] = dev_vecOut[0];
            dev_vecOut[0].x = 0;
            dev_vecOut[0].y = 0;
            numInputElements = subVecSize;
        }
    }

    hipfftHandle plan;
    cufftStatus = hipfftPlan1d(&plan, fftSize, HIPFFT_C2C, fftCount);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftPlan1d failed. Error code %d!\n", cufftStatus);
        return hipErrorUnknown;
    }

    cufftStatus = hipfftExecC2C(plan, dev_result,
        dev_result, HIPFFT_FORWARD);
    if (cufftStatus != HIPFFT_SUCCESS) {
        fprintf(stderr, "hipfftExecC2C failed. Error code %d!\n", cufftStatus);
        return hipErrorUnknown;
    }



    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_result), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!\n");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_filterTaps);
    hipFree(dev_subVec);
    hipFree(dev_result);
    hipFree(dev_vecOut);

    return cudaStatus;
}


__device__ hipfftComplex operator + (hipfftComplex const& a, hipfftComplex const& b) {
    hipfftComplex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

void readVectorFromFile(const char* fileName, float* result, int len) {
    FILE* file;
    file = fopen(fileName, "r");
    if (file == NULL) {
        printf("Error reading file!\n");
        return;
    }
    for (int m = 0; m < len; ++m) {
        fscanf(file, "%f ", &result[m]);
    }
    fclose(file);
}

void writeResultToFile(const char* fileName, float* result, int x, int y) {
    FILE* file;
    file = fopen(fileName, "w");

    int  n = 0;
    for (int l = 0; l < y; ++l) {
        for (int i = 0; i < x; ++i) {
            fprintf(file, "%f ", result[n]);
            n++;
        }
        fprintf(file, "\n");
    }
    fclose(file);
}
