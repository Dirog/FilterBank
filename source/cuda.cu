#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include ""
#include <iostream>
using namespace std;

#define DIM 1024

hipError_t execute(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, unsigned fftSize, unsigned step, float* result, unsigned resultLen);
void readVectorFromFile(char* fileName, float* result, int len);
void writeResultToFile(char* fileName, float* result, int x, int y);
void show_vector(float* vect, int size);
void show_complexVector(hipfftComplex* vect, int size);
__device__ hipfftComplex operator + (hipfftComplex const& a, hipfftComplex const& b);


__global__ void multiply(hipfftComplex* vect1, float* vect2, hipfftComplex* vecOut, int size, int step, int stepIndx)
{
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < size / step) {
        int indx = step * i + stepIndx;
        vecOut[i].x = vect1[indx].x * vect2[indx];
        vecOut[i].y = vect1[indx].y * vect2[indx];
    }
}

__global__ void average(hipfftComplex* vect, hipfftComplex* vecOut, int size, int fftSize, int stepIndx)
{
    __shared__ hipfftComplex block[DIM];
    unsigned int globalIndex = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int i = threadIdx.x;

    if (globalIndex < size) {
        block[i] = vect[globalIndex];
    }
    else {
        block[i].x = 0;
        block[i].y = 0;
    }

    __syncthreads();

    for (unsigned int j = blockDim.x / 2; j > 0; j >>= 1)
    {
        if (i < j)
            block[i] = block[i] + block[i + j];

        __syncthreads();
    }

    if (i == 0)
        vecOut[blockIdx.x] = block[0];

}


int main() {
    const int signalLen = 1024*8*2;
    const int filterLen = 512;
    const int fftSize = filterLen / 64;
    const int step = 128;
    int fftCount = ((signalLen / 2 - filterLen) / (step)) + 1;
    const int resultLen = 2 * fftSize * fftCount;
    float* result = new float[resultLen];

    float inSignal[signalLen];
    float filterTaps[filterLen];

    readVectorFromFile("signal", inSignal, signalLen);
    readVectorFromFile("taps", filterTaps, filterLen);

    hipError_t cudaStatus;
    cudaStatus = execute(inSignal, signalLen, filterTaps, filterLen, fftSize, step, result, resultLen);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Execution failed");
        return -1;
    }

    writeResultToFile("result", result, 2 * fftSize, fftCount);

    return 0;
}

hipError_t execute(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, const unsigned fftSize, unsigned step, float* result, unsigned resultLen)
{
    int threadsPerBlock = DIM;

    float* dev_inSignal;
    float* dev_filterTaps;
    hipfftComplex* dev_result;
    hipfftComplex* dev_subVec;
    hipfftComplex* dev_vecOut;
    hipfftComplex* dev_fftVec;

    int fftCount = ((signalLen / 2 - filterLen) / step) + 1;
    int subVecSize = filterLen / fftSize;
    
    hipError_t cudaStatus;
    hipfftResult cufftStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_inSignal, signalLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    cudaStatus = hipMallocManaged((float**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_filterTaps, filterLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((float**)&dev_subVec, filterLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    cudaStatus = hipMallocManaged((float**)&dev_vecOut, subVecSize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    cudaStatus = hipMallocManaged((float**)&dev_fftVec, fftSize * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_inSignal, inSignal, signalLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_filterTaps, filterTaps, filterLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return cudaStatus;
    }

    hipfftComplex* dev_inComplexSignal = reinterpret_cast<hipfftComplex*>(dev_inSignal);
    int numInputElements = subVecSize;
    int numOutputElements;
    
    for (int batchIndx = 0; batchIndx < fftCount; batchIndx++)
    {
        for (int stepIndx = 0; stepIndx < fftSize; stepIndx++) {
            multiply << <256, threadsPerBlock >> > (dev_inComplexSignal + batchIndx*step, dev_filterTaps, dev_subVec, filterLen, fftSize, stepIndx);
            do
            {
                numOutputElements = numInputElements / (threadsPerBlock);
                if (numInputElements % (threadsPerBlock)) {
                    numOutputElements++;
                }

                average << < numOutputElements, threadsPerBlock >> > (dev_subVec, dev_vecOut, numInputElements, fftSize, stepIndx);
                numInputElements = numOutputElements;
                if (numOutputElements > 1) {
                    average << < numOutputElements, threadsPerBlock >> > (dev_vecOut, dev_subVec, numInputElements, fftSize, stepIndx);
                }

            } while (numOutputElements > 1);

            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess) {
                fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
                return cudaStatus;
            }

            dev_fftVec[stepIndx] = dev_vecOut[0];
            dev_result[stepIndx + batchIndx*fftSize] = dev_vecOut[0];
            dev_vecOut[0].x = 0;
            dev_vecOut[0].y = 0;
            numInputElements = subVecSize;
        }

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
            return cudaStatus;
        }

        //show_complexVector(dev_fftVec, fftSize);

    }

    hipfftHandle plan;
    cufftStatus = hipfftPlan1d(&plan, fftSize, HIPFFT_C2C, fftCount);
    if (cufftStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    cufftStatus = hipfftExecC2C(plan, dev_result,
        dev_result, HIPFFT_FORWARD);
    if (cufftStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }



    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return cudaStatus;
    }


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_result), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return cudaStatus;
    }

    hipFree(dev_inSignal);
    hipFree(dev_filterTaps);
    hipFree(dev_subVec);
    hipFree(dev_result);
    hipFree(dev_vecOut);
    hipFree(dev_fftVec);

    return cudaStatus;
}


__device__ hipfftComplex operator + (hipfftComplex const& a, hipfftComplex const& b) {
    hipfftComplex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

void readVectorFromFile(char* fileName, float* result, int len) {
    FILE* signal_file;
    signal_file = fopen(fileName, "r");
    if (signal_file == NULL) {
        printf("Error reading file!");
        return;
    }
    for (int m = 0; m < len; ++m) {
        fscanf(signal_file, "%f ", &result[m]);
    }
    fclose(signal_file);
}

void writeResultToFile(char* fileName, float* result, int x, int y) {
    FILE* file;
    file = fopen(fileName, "w");

    int  n = 0;
    for (int l = 0; l < y; ++l) {
        for (int i = 0; i < x; ++i) {
            fprintf(file, "%f ", result[n]);
            n++;
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

void show_vector(float* vect, int size)
{
    for (int i = 0; i < size; i++)
        cout << vect[i] << " ";
    cout << endl;
}

void show_complexVector(hipfftComplex* vect, int size)
{
    for (int i = 0; i < size; i++) {
        cout << vect[i].x;
        if (vect[i].y >= 0) {
            cout << "+" << vect[i].y << "i";
        }
        else {
            cout << vect[i].y << "i";
        }
        cout << endl;
    }

    cout << endl;
}