#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>





hipError_t multiplyAndAverageWithCuda(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, unsigned fftSize, float* result);

__global__ void multiplyKernel(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, unsigned fftSize, float* result, unsigned resultLen) {
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < signalLen)
    {
        int j = i % filterLen;
        result[2 * i] = inSignal[2 * i] * filterTaps[j];
        result[2 * i + 1] = inSignal[2 * i + 1] * filterTaps[j];
    }


}


int main() {
    const int arrSize = 2 * 16;
    const int hSize = 8;
    const int fftSize = 8;
    const int resultLen = fftSize * (arrSize / hSize);
    float result[resultLen];
    float inSignal[arrSize] = { 1, 1, 2, 2, 3, 3, 4, 4, 5, 5, 6, 6, 7, 7, 8, 8, 9, 9, 10, 10, 11, 11, 12, 12, 13, 13, 14, 14, 15, 15, 16, 16 };
    float h[hSize] = { 1, 2, 3, 4, 5, 6, 7, 8 };

    multiplyAndAverageWithCuda(inSignal, arrSize, h, hSize, fftSize, result);
   
    return 0;
}

hipError_t multiplyAndAverageWithCuda(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, unsigned fftSize, float* result)
{
    float* dev_inSignal = 0;
    float* dev_result = 0;
    float* dev_filterTaps = 0;
    const int resultLen = fftSize * (signalLen / filterLen);
    hipError_t cudaStatus;
    hipfftResult cufftStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_inSignal, signalLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_filterTaps, filterLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_inSignal, inSignal, signalLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_filterTaps, filterTaps, filterLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    multiplyKernel <<<256, 1024>>> (dev_inSignal, signalLen, dev_filterTaps, filterLen, fftSize, dev_result, resultLen);


    hipfftHandle plan;
    cufftStatus = hipfftPlan1d(&plan, fftSize, HIPFFT_C2C, signalLen / fftSize);
    if (cufftStatus != hipSuccess) {
        fprintf(stderr, "cufftPlan failed!");
        goto Error;
    }

    cufftStatus = hipfftExecC2C(plan, reinterpret_cast<hipfftComplex*>(dev_result),
                    reinterpret_cast<hipfftComplex*>(dev_result),
                    HIPFFT_FORWARD);
    if (cufftStatus != hipSuccess) {
        fprintf(stderr, "cufftExec failed!");
        goto Error;
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(result, dev_result, resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(inSignal, dev_inSignal, signalLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_inSignal);
    hipFree(dev_filterTaps);

    return cudaStatus;
}
