#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


__global__ void multiplyKernel(float * inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, float * result){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = 0;
    if (i < signalLen) {
        result[2*i] = inSignal[2*i] * filterTaps[j]; //Re
        result[2*i + 1] = inSignal[2*i + 1] * filterTaps[j]; //Im
        j++;
        if (j >= filterLen) {
            j = 0;
        }
    }
}

hipError_t multiplyWithCuda(float * inSignal, unsigned signalLen, float* dev_filterTaps, unsigned filterLen, float * result)
{
    float * dev_inSignal; 
    float *dev_result;
    unsigned resultLen = signalLen;
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_inSignal, signalLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_inSignal, inSignal, signalLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }



    // int N = 256;
    // dim3 threadsPerBlock(N, N);
    // dim3 numBlocks;    
    // multiplyKernel<<<numBlocks, threadsPerBlock>>>(dev_inSignal, signalLen, dev_filterTaps, filterLen, dev_result);


    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        goto Error;
    }

//     cudaStatus = hipMemcpy(result, dev_result, resultLen * sizeof(float), hipMemcpyDeviceToHost);
//     if (cudaStatus != hipSuccess) {
//         fprintf(stderr, "hipMemcpy failed!");
//         goto Error;
//     }

// Error:
//     hipFree(dev_inSignal);
//     hipFree(dev_result);
    
    return cudaStatus;
}
