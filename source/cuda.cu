#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include ""



hipError_t execute(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, unsigned fftSize, unsigned step, float* result, unsigned resultLen);
void readVectorFromFile(char* fileName, float* result, int len);
void writeResultToFile(char* fileName, float* result, int x, int y);

__global__ void multiplyKernel(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, unsigned fftSize, float* result, unsigned resultLen) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < filterLen)
    {
        result[2 * i] = inSignal[2 * i] * filterTaps[i];
        result[2 * i + 1] = inSignal[2 * i + 1] * filterTaps[i];
    }


}


int main() {
    const int signalLen = 16 * 2;
    const int filterLen = 8;
    const int fftSize = filterLen;
    const int step = 4;
    const int fftCount = ((signalLen / 2 - filterLen / 2) / (step)) - 1;
    const int resultLen = 2 * fftSize * fftCount;
    float result[resultLen];

    float inSignal[signalLen];
    float filterTaps[filterLen];

    readVectorFromFile("sine", inSignal, signalLen);
    readVectorFromFile("taps", filterTaps, filterLen);

    execute(inSignal, signalLen, filterTaps, filterLen, fftSize, step, result, resultLen);

    writeResultToFile("result", result, 2 * fftSize, fftCount);

    /*for (int i = 0; i < resultLen / 2; i++) {
        printf("%f", result[2 * i]);
        if (result[2 * i + 1] >= 0)
            printf(" + %fi\n", result[2 * i + 1]);
        else
            printf(" %fi\n", result[2 * i + 1]);
    }*/

    return 0;
}

hipError_t execute(float* inSignal, unsigned signalLen, float* filterTaps, unsigned filterLen, unsigned fftSize, unsigned step, float* result, unsigned resultLen)
{
    float* dev_inSignal = 0;
    float* dev_filterTaps = 0;
    int fftCount = ((signalLen / 2 - filterLen / 2) / (step)) - 1;
    float* dev_result = 0;
    float* dev_mlpResult = 0;
    int mlpResultLen = filterLen;
    hipError_t cudaStatus;
    hipfftResult cufftStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_inSignal, signalLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_result, resultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_mlpResult, mlpResultLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_filterTaps, filterLen * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_inSignal, inSignal, signalLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    cudaStatus = hipMemcpy(dev_filterTaps, filterTaps, filterLen * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


    cuffthandle plan;
    cufftstatus = cufftplan1d(&plan, 2 * fftsize, cufft_c2c, 1);
    if (cufftstatus != cudasuccess) {
        fprintf(stderr, "cufftplan failed!");
        goto error;
    }

    for (int i = 0; i < fftCount; i++)
    {
        multiplyKernel<<<256, 256>>> (dev_inSignal, signalLen, dev_filterTaps, filterLen, fftSize, dev_mlpResult, mlpResultLen);

        cufftStatus = hipfftExecC2C(plan, reinterpret_cast<hipfftComplex*>(dev_mlpResult),
            reinterpret_cast<hipfftComplex*>(dev_result + i * 2 * fftSize),
            HIPFFT_FORWARD);
        if (cufftStatus != hipSuccess) {
            fprintf(stderr, "cufftExec failed!");
            goto Error;
        }
    }

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Kernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(result, reinterpret_cast<float*>(dev_result), resultLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(inSignal, dev_inSignal, signalLen * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_inSignal);
    hipFree(dev_filterTaps);

    return cudaStatus;
}


void readVectorFromFile(char* fileName, float* result, int len) {
    FILE* signal_file;
    signal_file = fopen(fileName, "r");
    if (signal_file == NULL) {
        printf("Error reading file!");
        return;
    }
    for (int m = 0; m < len; ++m) {
        fscanf(signal_file, "%f ", &result[m]);
    }
    fclose(signal_file);
}

void writeResultToFile(char* fileName, float* result, int x, int y) {
    FILE* file;
    file = fopen(fileName, "w");

    int  n = 0;
    for (int l = 0; l < y; ++l) {
        for (int i = 0; i < x; ++i) {
            fprintf(file, "%f ", result[n]);
            n++;
        }
        fprintf(file, "\n");
    }
    fclose(file);
}